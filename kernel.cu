
#include <hip/hip_runtime.h>
/**
 * @brief Kernel function to solve the problem on the GPU.
 *
 * This function is expected to perform computations on the GPU using the
 * provided input data. It processes the changes and updates the account and sum
 * arrays accordingly.
 *
 * @param changes Pointer to an array of changes to be applied. It is a 2D array
 * where each "row" represents a client and each "column" represents a period.
 * Each element represents the change in the account balance for a client in a
 * period.
 * @param account Pointer to an array representing the account balances. It is a
 * 2D array where each "row" represents a client and each "column" represents a
 * period. Each element represents the account balance for a client in a period.
 * @param sum Pointer to an array representing the sum of account balances for
 * each period. It is a 1D array where each element represents the sum of
 * account balances for a period.
 * @param clients The number of clients. It is the number of rows in the
 * changes, account, and sum arrays.
 * @param periods The number of periods. It is the number of columns in the
 * changes, account, and sum arrays.
 *
 * The function does not return any value. It modifies the account and sum
 * arrays in place.
 */

/*/////////////Some consts////////////////*/
// TODO dicide on specific consts
#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5
#define MAX_THREADS_PER_BLOCK 512
#define ELEMENTS_PER_BLOCK MAX_THREADS_PER_BLOCK * 2

// TODO get more info about this
// There were two BCAO optimisations in the paper - this one is fastest
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)
/*////////////////////////////////////////*/

/*////////////////TODO my stuff///////////*/
void cp_from_device_and_print(int *d_arr, int length) {
  int *arr = (int *)malloc(length * sizeof(int));
  hipMemcpy(arr, d_arr, length * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < length; i++) {
    printf("%d ", arr[i]);
  }
  printf("\n");
  free(arr);
}
/*////////////////////////////////////////*/

/*//////////////cuda stuff////////////////*/
__global__ void add(int *output, int length, int *n) {
  int blockID = blockIdx.x;
  int threadID = threadIdx.x;
  int blockOffset = blockID * length;

  output[blockOffset + threadID] += n[blockID];
}

/*////////////////////////////////////////*/

/*////////cuda prefix sum/////////////////*/

// Kernel function to perform a large prefix sum (scan) operation

__global__ void scan(int *output, int *input, int n, int *sums) {
  // Shared memory for temporary storage
  extern __shared__ int temp[];

  // Calculate block and thread indices
  int blockID = blockIdx.x;
  int threadID = threadIdx.x;
  int blockOffset = blockID * n;

  // Calculate indices for the elements this thread will work on
  int ai = threadID;
  int bi = threadID + (n / 2);

  // Calculate bank offsets to avoid shared memory bank conflicts
  int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
  int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

  // Load input elements into shared memory with bank conflict avoidance
  temp[ai + bankOffsetA] = input[blockOffset + ai];
  temp[bi + bankOffsetB] = input[blockOffset + bi];

  // Offset for the reduction step
  int offset = 1;

  // Build sum in place up the tree - up-sweep phase
  for (int d = n >> 1; d > 0; d >>= 1) {
    // Synchronize threads to ensure all data is loaded
    __syncthreads();
    if (threadID < d) {
      // Calculate indices for the reduction step
      int ai = offset * (2 * threadID + 1) - 1;
      int bi = offset * (2 * threadID + 2) - 1;

      // Apply bank offsets
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      // Perform the reduction
      temp[bi] += temp[ai];
    }
    // Double the offset for the next level of the tree
    offset *= 2;
  }

  // Synchronize threads before moving to the next phase
  __syncthreads();

  // Save the total sum of this block to the sums array and clear the last
  // element
  if (threadID == 0) {
    sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
    temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
  }

  // Traverse down the tree and build the scan - down-sweep phase
  for (int d = 1; d < n; d *= 2) {
    // Halve the offset for the next level of the tree
    offset >>= 1;
    // Synchronize threads before the next step
    __syncthreads();
    if (threadID < d) {
      // Calculate indices for the scan step
      int ai = offset * (2 * threadID + 1) - 1;
      int bi = offset * (2 * threadID + 2) - 1;

      // Apply bank offsets
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      // Perform the scan operation
      int t = temp[ai];
      temp[ai] = temp[bi];
      temp[bi] += t;
    }
  }

  // Synchronize threads before writing the output
  __syncthreads();

  // Write the results to the output array
  output[blockOffset + ai] = temp[ai + bankOffsetA] + input[blockOffset + ai];
  output[blockOffset + bi] = temp[bi + bankOffsetB] + input[blockOffset + bi];
}

/*///////////////////////////////////*/

/*///////////scan funcs//////////////*/

/*///////////////////////////////////*/

__global__ void calc_single_client_acc(int *account, int *changes, int clients,
                                       int periods) {
  int client = blockIdx.x * blockDim.x + threadIdx.x;

  if (periods <= ELEMENTS_PER_BLOCK) {
    int threads = periods / 2;
    int *sums;
    hipMalloc((void **)&sums, sizeof(sums[0])); // TODO remove this
    scan<<<1, threads, periods * sizeof(int)>>>(account, changes, periods,
                                                sums);
    hipFree(sums);
  } else {
  }
}

void calc_clients_accounts(int *account, int *changes, int clients,
                           int periods) {
  int clients_per_block = MAX_THREADS_PER_BLOCK;
  while (!(clients % clients_per_block == 0 && clients_per_block > 128)) {
    clients_per_block -= 2;
  }
  int blocks = clients / clients_per_block;

  calc_single_client_acc<<<blocks, clients_per_block>>>(account, changes,
                                                        clients, periods);
}

void calc_accounts(int *account, int *changes, int clients, int periods) {
  int client_num_of_periods = periods;
  if (client_num_of_periods > ELEMENTS_PER_BLOCK) {
    // perform a large scan (meaning more than one block per client)
    // to distinguish the clients blocks from each other we define 2Dim grid
    // where the x axis is the client and the y axis is the block
    int remainder = client_num_of_periods % ELEMENTS_PER_BLOCK;
    // if we can fit the clients periods exactly in the blocks with max
    // elements per block
    int blocks_per_client = client_num_of_periods / ELEMENTS_PER_BLOCK;
    dim3 numBlocks(clients, blocks_per_client);
    dim3 threadsPerBlock(ELEMENTS_PER_BLOCK);
    // else
    // option 1
    if (remainder != 0) {
      // find a number of elements per block (meaning must be <= than
      // ELEMENTS_PER_BLOCK) that can divide the number of periods that way we
      // ensure that each block is full and each client has the same number of
      // blocks

      // find the number of elements per block
      int elements_per_block = 0;
      for (int i = ELEMENTS_PER_BLOCK; i > 0; i--) {
        if (client_num_of_periods % i == 0) {
          elements_per_block = i;
          break;
        }
      }

      // find the number of blocks per client
      blocks_per_client = client_num_of_periods / elements_per_block;
      dim3 numBlocks(clients, blocks_per_client);
      dim3 threadsPerBlock(elements_per_block);
    }

    // TODO scan large <<<numBlocks, threadsPerBlock>>>
  } else {
    // only need one block to scan each client
    // so here we use a block for each client and set the number of threads to
    // the number of periods
    dim3 numBlocks(clients);
    dim3 threadsPerBlock(client_num_of_periods);

    // TODO scan small <<<numBlocks, threadsPerBlock>>>
    int *sums;
    hipMalloc((void **)&sums, sizeof(sums[0]));
    scan<<<numBlocks, threadsPerBlock>>>(account, changes, periods, sums);
  }
}

__global__ void copy_first_column(int *output, int *input, int clients,
                                  int periods) {
  int threadID = threadIdx.x;

  output[threadID] = input[threadID * clients];
}

void solveGPU(int *changes, int *account, int *sum, int clients, int periods) {
  // copy first collumn of changes to a new array, all is on gpu
  int *n;
  hipMalloc((void **)&n, periods * sizeof(n[0]));
  copy_first_column<<<1, periods>>>(n, changes, clients, periods);
  printf("n: ");
  cp_from_device_and_print(n, periods);

  int *out;
  hipMalloc((void **)&out, periods * sizeof(out[0]));
  hipMemset(out, 0, periods * sizeof(out[0]));

  int *sums;
  hipMalloc((void **)&sums, sizeof(sums[0]));
  hipMemset(out, 0, sizeof(out[0]));

  cp_from_device_and_print(out, periods);

  int shared_mem_size = periods * sizeof(int);
  scan<<<1, periods / 2, shared_mem_size>>>(out, n, periods, sums);
  cp_from_device_and_print(out, periods);
  printf("sums: ");
  cp_from_device_and_print(sums, 1);
}
